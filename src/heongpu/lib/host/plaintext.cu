// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "plaintext.cuh"

namespace heongpu
{
    __host__ Plaintext::Plaintext() {}

    __host__ Plaintext::Plaintext(Parameters& context, hipStream_t stream)
    {
        scheme_ = context.scheme_;
        switch (static_cast<int>(context.scheme_))
        {
            case 1: // BFV
                plain_size_ = context.n;
                depth_ = 0;
                scale_ = 0;
                in_ntt_domain_ = false;
                break;
            case 2: // CKKS
                plain_size_ = context.n * context.Q_size; // n
                depth_ = 0;
                scale_ = 0;
                in_ntt_domain_ = true;
                break;
            default:
                break;
        }

        locations_ = DeviceVector<Data>(plain_size_, stream);
    }

    __host__ Plaintext::Plaintext(const std::vector<Data>& plain,
                                  Parameters& context, hipStream_t stream)
    {
        scheme_ = context.scheme_;
        switch (static_cast<int>(context.scheme_))
        {
            case 1: // BFV
                plain_size_ = context.n;
                depth_ = 0;
                scale_ = 0;
                in_ntt_domain_ = false;

                if (!(plain.size() == plain_size_))
                {
                    throw std::invalid_argument(
                        "Plaintext size should be valid!");
                }

                break;
            case 2: // CKKS
                plain_size_ = context.n * context.Q_size; // n
                depth_ = 0;
                scale_ = 0;
                in_ntt_domain_ = true;

                if (!(plain.size() == plain_size_))
                {
                    throw std::invalid_argument(
                        "Plaintext size should be valid!");
                }

                break;
            default:
                break;
        }

        locations_ = DeviceVector<Data>(plain, stream);
    }

    __host__ Plaintext::Plaintext(const HostVector<Data>& plain,
                                  Parameters& context, hipStream_t stream)
    {
        scheme_ = context.scheme_;
        switch (static_cast<int>(context.scheme_))
        {
            case 1: // BFV
                plain_size_ = context.n;
                depth_ = 0;
                scale_ = 0;
                in_ntt_domain_ = false;

                if (!(plain.size() == plain_size_))
                {
                    throw std::invalid_argument(
                        "Plaintext size should be valid!");
                }

                break;
            case 2: // CKKS
                plain_size_ = context.n * context.Q_size; // n
                depth_ = 0;
                scale_ = 0;
                in_ntt_domain_ = true;

                if (!(plain.size() == plain_size_))
                {
                    throw std::invalid_argument(
                        "Plaintext size should be valid!");
                }

                break;
            default:
                break;
        }

        locations_ = DeviceVector<Data>(plain, stream);
    }

    Data* Plaintext::data()
    {
        return locations_.data();
    }

    void Plaintext::device_to_host(std::vector<Data>& plain,
                                   hipStream_t stream)
    {
        if (plain.size() < plain_size_)
        {
            plain.resize(plain_size_);
        }

        hipMemcpyAsync(plain.data(), locations_.data(),
                        plain_size_ * sizeof(Data), hipMemcpyDeviceToHost,
                        stream);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    void Plaintext::host_to_device(std::vector<Data>& plain,
                                   hipStream_t stream)
    {
        if (!(plain.size() == plain_size_))
        {
            throw std::invalid_argument("Plaintext size should be valid!");
        }

        hipMemcpyAsync(locations_.data(), plain.data(),
                        plain_size_ * sizeof(Data), hipMemcpyHostToDevice,
                        stream);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    void Plaintext::device_to_host(HostVector<Data>& plain, hipStream_t stream)
    {
        if (plain.size() < plain_size_)
        {
            plain.resize(plain_size_);
        }

        hipMemcpyAsync(plain.data(), locations_.data(),
                        plain_size_ * sizeof(Data), hipMemcpyDeviceToHost,
                        stream);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    void Plaintext::host_to_device(HostVector<Data>& plain, hipStream_t stream)
    {
        if (!(plain.size() == plain_size_))
        {
            throw std::invalid_argument("Plaintext size should be valid!");
        }

        hipMemcpyAsync(locations_.data(), plain.data(),
                        plain_size_ * sizeof(Data), hipMemcpyHostToDevice,
                        stream);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

} // namespace heongpu