﻿// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "decryptor.cuh"

namespace heongpu
{
    __host__ HEDecryptor::HEDecryptor(Parameters& context,
                                      Secretkey& secret_key)
    {
        scheme = context.scheme_;

        secret_key_ = secret_key.data();

        n = context.n;
        n_power = context.n_power;

        decomp_mod_count_ = context.Q_size;

        modulus_ = context.modulus_;

        ntt_table_ = context.ntt_table_;
        intt_table_ = context.intt_table_;

        n_inverse_ = context.n_inverse_;

        if (scheme == scheme_type::bfv)
        {
            plain_modulus_ = context.plain_modulus_;

            gamma_ = context.gamma_;

            Qi_t_ = context.Qi_t_;

            Qi_gamma_ = context.Qi_gamma_;

            Qi_inverse_ = context.Qi_inverse_;

            mulq_inv_t_ = context.mulq_inv_t_;

            mulq_inv_gamma_ = context.mulq_inv_gamma_;

            inv_gamma_ = context.inv_gamma_;

            // Noise budget calculation

            Mi_ = context.Mi_;
            Mi_inv_ = context.Mi_inv_;
            upper_half_threshold_ = context.upper_half_threshold_;
            decryption_modulus_ = context.decryption_modulus_;
            total_bit_count_ = context.total_bit_count_;

            temp_memory_ = DeviceVector<Data>(n * decomp_mod_count_);
            temp_memory2_ = DeviceVector<Data>(2 * n * decomp_mod_count_);

            // max_norm_memory_ =
            //     (Data*) malloc(n * decomp_mod_count_ * sizeof(Data));
            max_norm_memory_.resize(n * decomp_mod_count_);
        }
    }

    __host__ void HEDecryptor::decrypt_bfv(Plaintext& plaintext,
                                           Ciphertext& ciphertext)
    {
        Data* ct0 = ciphertext.data();
        Data* ct1 = ciphertext.data() + (decomp_mod_count_ << n_power);

        Data* ct0_temp = temp_memory2_.data();
        Data* ct1_temp = temp_memory2_.data() + (decomp_mod_count_ << n_power);

        ntt_rns_configuration cfg_ntt = {.n_power = n_power,
                                         .ntt_type = FORWARD,
                                         .reduction_poly =
                                             ReductionPolynomial::X_N_plus,
                                         .zero_padding = false,
                                         .stream = 0};
        if (!ciphertext.in_ntt_domain_)
        {
            GPU_NTT(ct1, ct1_temp, ntt_table_->data(), modulus_->data(),
                    cfg_ntt, decomp_mod_count_, decomp_mod_count_);

            sk_multiplication<<<dim3((n >> 8), decomp_mod_count_, 1), 256>>>(
                ct1_temp, secret_key_, ct1_temp, modulus_->data(), n_power,
                decomp_mod_count_);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            sk_multiplication<<<dim3((n >> 8), decomp_mod_count_, 1), 256>>>(
                ct1, secret_key_, ct1_temp, modulus_->data(), n_power,
                decomp_mod_count_);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        ntt_rns_configuration cfg_intt = {.n_power = n_power,
                                          .ntt_type = INVERSE,
                                          .reduction_poly =
                                              ReductionPolynomial::X_N_plus,
                                          .zero_padding = false,
                                          .mod_inverse = n_inverse_->data(),
                                          .stream = 0};

        if (ciphertext.in_ntt_domain_)
        {
            // TODO: merge these NTTs
            GPU_NTT(ct0, ct0_temp, intt_table_->data(), modulus_->data(),
                    cfg_intt, decomp_mod_count_, decomp_mod_count_);

            GPU_NTT_Inplace(ct1_temp, intt_table_->data(), modulus_->data(),
                            cfg_intt, decomp_mod_count_, decomp_mod_count_);

            ct0 = ct0_temp;
        }
        else
        {
            GPU_NTT_Inplace(ct1_temp, intt_table_->data(), modulus_->data(),
                            cfg_intt, decomp_mod_count_, decomp_mod_count_);
        }

        decryption_kernel<<<dim3((n >> 8), 1, 1), 256>>>(
            ct0, ct1_temp, plaintext.data(), modulus_->data(), plain_modulus_,
            gamma_, Qi_t_->data(), Qi_gamma_->data(), Qi_inverse_->data(),
            mulq_inv_t_, mulq_inv_gamma_, inv_gamma_, n_power,
            decomp_mod_count_);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ void HEDecryptor::decryptx3_bfv(Plaintext& plaintext,
                                             Ciphertext& ciphertext)
    {
        Data* ct0 = ciphertext.data();
        Data* ct1 = ciphertext.data() + (decomp_mod_count_ << n_power);
        Data* ct2 = ciphertext.data() + (decomp_mod_count_ << (n_power + 1));

        ntt_rns_configuration cfg_ntt = {.n_power = n_power,
                                         .ntt_type = FORWARD,
                                         .reduction_poly =
                                             ReductionPolynomial::X_N_plus,
                                         .zero_padding = false,
                                         .stream = 0};

        GPU_NTT_Inplace(ct1, ntt_table_->data(), modulus_->data(), cfg_ntt,
                        2 * decomp_mod_count_, decomp_mod_count_);

        sk_multiplicationx3<<<dim3((n >> 8), decomp_mod_count_, 1), 256>>>(
            ct1, secret_key_, modulus_->data(), n_power, decomp_mod_count_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        ntt_rns_configuration cfg_intt = {.n_power = n_power,
                                          .ntt_type = INVERSE,
                                          .reduction_poly =
                                              ReductionPolynomial::X_N_plus,
                                          .zero_padding = false,
                                          .mod_inverse = n_inverse_->data(),
                                          .stream = 0};

        GPU_NTT_Inplace(ct1, intt_table_->data(), modulus_->data(), cfg_intt,
                        2 * decomp_mod_count_, decomp_mod_count_);

        decryption_kernelx3<<<dim3((n >> 8), 1, 1), 256>>>(
            ct0, ct1, ct2, plaintext.data(), modulus_->data(), plain_modulus_,
            gamma_, Qi_t_->data(), Qi_gamma_->data(), Qi_inverse_->data(),
            mulq_inv_t_, mulq_inv_gamma_, inv_gamma_, n_power,
            decomp_mod_count_);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ int HEDecryptor::noise_budget_calculation(Ciphertext& ciphertext)
    {
        Data* ct0 = ciphertext.data();
        Data* ct1 = ciphertext.data() + (decomp_mod_count_ << n_power);

        ntt_rns_configuration cfg_ntt = {.n_power = n_power,
                                         .ntt_type = FORWARD,
                                         .reduction_poly =
                                             ReductionPolynomial::X_N_plus,
                                         .zero_padding = false,
                                         .stream = 0};

        GPU_NTT(ct1, temp_memory_.data(), ntt_table_->data(), modulus_->data(),
                cfg_ntt, decomp_mod_count_, decomp_mod_count_);

        sk_multiplication<<<dim3((n >> 8), decomp_mod_count_, 1), 256>>>(
            temp_memory_.data(), secret_key_, temp_memory_.data(),
            modulus_->data(), n_power, decomp_mod_count_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        ntt_rns_configuration cfg_intt = {.n_power = n_power,
                                          .ntt_type = INVERSE,
                                          .reduction_poly =
                                              ReductionPolynomial::X_N_plus,
                                          .zero_padding = false,
                                          .mod_inverse = n_inverse_->data(),
                                          .stream = 0};

        GPU_NTT_Inplace(temp_memory_.data(), intt_table_->data(),
                        modulus_->data(), cfg_intt, decomp_mod_count_,
                        decomp_mod_count_);

        coeff_multadd<<<dim3((n >> 8), decomp_mod_count_, 1), 256>>>(
            ct0, temp_memory_.data(), temp_memory_.data(), plain_modulus_,
            modulus_->data(), n_power, decomp_mod_count_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        compose_kernel<<<dim3((n >> 8), 1, 1), 256>>>(
            temp_memory_.data(), temp_memory_.data(), modulus_->data(),
            Mi_inv_->data(), Mi_->data(), decryption_modulus_->data(),
            decomp_mod_count_, n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        find_max_norm_kernel<<<1, 512, sizeof(Data) * 512>>>(
            temp_memory_.data(), temp_memory_.data(),
            upper_half_threshold_->data(), decryption_modulus_->data(),
            decomp_mod_count_,
            n_power); // TODO: merge with above kernel if possible
        HEONGPU_CUDA_CHECK(hipGetLastError());

        hipMemcpy(max_norm_memory_.data(), temp_memory_.data(),
                   decomp_mod_count_ * sizeof(Data), hipMemcpyDeviceToHost);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        return total_bit_count_ -
               calculate_big_integer_bit_count(max_norm_memory_.data(),
                                               decomp_mod_count_) -
               1;
    }

    __host__ void HEDecryptor::decrypt_ckks(Plaintext& plaintext,
                                            Ciphertext& ciphertext)
    {
        int current_decomp_count = decomp_mod_count_ - ciphertext.depth_;

        sk_multiplication_ckks<<<dim3((n >> 8), current_decomp_count, 1),
                                 256>>>(ciphertext.data(), plaintext.data(),
                                        secret_key_, modulus_->data(), n_power,
                                        current_decomp_count);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        plaintext.scale_ = ciphertext.scale_;
        plaintext.depth_ = ciphertext.depth_;
    }

} // namespace heongpu